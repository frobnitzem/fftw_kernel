#include "hip/hip_runtime.h"
#include "rfft.h"

__global__ void r2hc_32(const R I, R ro, R io) {
     DK(KP831469612, +0.831469612302545237078788377617905756738560812);
     DK(KP668178637, +0.668178637919298919997757686523080761552472251);
     DK(KP980785280, +0.980785280403230449126182236134239036973933731);
     DK(KP198912367, +0.198912367379658006911597622644676228597850501);
     DK(KP923879532, +0.923879532511286756128183189396788286822416626);
     DK(KP707106781, +0.707106781186547524400844362104849039284835938);
     DK(KP414213562, +0.414213562373095048801688724209698078569671875);
	  E T1x, T1M, T1I, T1E, T1J, T1H;
	  {
	       E Tv, T1h, T7, T2b, Te, T2n, Ty, T1i, T1l, TF, T2d, Tt, T1k, TC, T2c;
	       E Tm, T2j, T1Z, T2k, T22, TK, T1B, T19, T1C, T1e, TO, TV, T1T, TN, TP;
	       E T2g, T1S;
	       {
		    E TD, Tp, Tq, Tr;
		    {
			 E T1, T2, T4, T5;
			 T1 = I[0];
			 T2 = I[WS(is, 16)];
			 T4 = I[WS(is, 8)];
			 T5 = I[WS(is, 24)];
			 {
			      E Ta, Tw, Tx, Td, Tn, To;
			      {
				   E T8, T3, T6, T9, Tb, Tc;
				   T8 = I[WS(is, 4)];
				   Tv = T1 - T2;
				   T3 = T1 + T2;
				   T1h = T4 - T5;
				   T6 = T4 + T5;
				   T9 = I[WS(is, 20)];
				   Tb = I[WS(is, 28)];
				   Tc = I[WS(is, 12)];
				   T7 = T3 + T6;
				   T2b = T3 - T6;
				   Ta = T8 + T9;
				   Tw = T8 - T9;
				   Tx = Tb - Tc;
				   Td = Tb + Tc;
			      }
			      Tn = I[WS(is, 30)];
			      To = I[WS(is, 14)];
			      Te = Ta + Td;
			      T2n = Td - Ta;
			      Ty = Tw + Tx;
			      T1i = Tx - Tw;
			      TD = Tn - To;
			      Tp = Tn + To;
			      Tq = I[WS(is, 6)];
			      Tr = I[WS(is, 22)];
			 }
		    }
		    {
			 E Tj, TA, Ti, Tk;
			 {
			      E Tg, Th, TE, Ts;
			      Tg = I[WS(is, 2)];
			      Th = I[WS(is, 18)];
			      Tj = I[WS(is, 10)];
			      TE = Tq - Tr;
			      Ts = Tq + Tr;
			      TA = Tg - Th;
			      Ti = Tg + Th;
			      T1l = FNMS(KP414213562, TD, TE);
			      TF = FMA(KP414213562, TE, TD);
			      T2d = Tp - Ts;
			      Tt = Tp + Ts;
			      Tk = I[WS(is, 26)];
			 }
			 {
			      E T11, T15, T1c, T20, T14, T16, T1X, T1Y, T1Q, T1R;
			      {
				   E T1a, T1b, T12, T13;
				   {
					E TZ, T10, TB, Tl;
					TZ = I[WS(is, 31)];
					T10 = I[WS(is, 15)];
					T1a = I[WS(is, 23)];
					TB = Tj - Tk;
					Tl = Tj + Tk;
					T1X = TZ + T10;
					T11 = TZ - T10;
					T1k = FMA(KP414213562, TA, TB);
					TC = FNMS(KP414213562, TB, TA);
					T2c = Ti - Tl;
					Tm = Ti + Tl;
					T1b = I[WS(is, 7)];
				   }
				   T12 = I[WS(is, 3)];
				   T13 = I[WS(is, 19)];
				   T15 = I[WS(is, 27)];
				   T1Y = T1b + T1a;
				   T1c = T1a - T1b;
				   T20 = T12 + T13;
				   T14 = T12 - T13;
				   T16 = I[WS(is, 11)];
			      }
			      T2j = T1X - T1Y;
			      T1Z = T1X + T1Y;
			      {
				   E TT, TU, TL, TM;
				   {
					E TI, T21, T17, TJ, T18, T1d;
					TI = I[WS(is, 1)];
					T21 = T15 + T16;
					T17 = T15 - T16;
					TJ = I[WS(is, 17)];
					TT = I[WS(is, 9)];
					T2k = T21 - T20;
					T22 = T20 + T21;
					T18 = T14 + T17;
					T1d = T17 - T14;
					T1Q = TI + TJ;
					TK = TI - TJ;
					T1B = FNMS(KP707106781, T18, T11);
					T19 = FMA(KP707106781, T18, T11);
					T1C = FNMS(KP707106781, T1d, T1c);
					T1e = FMA(KP707106781, T1d, T1c);
					TU = I[WS(is, 25)];
				   }
				   TL = I[WS(is, 5)];
				   TM = I[WS(is, 21)];
				   TO = I[WS(is, 29)];
				   T1R = TT + TU;
				   TV = TT - TU;
				   T1T = TL + TM;
				   TN = TL - TM;
				   TP = I[WS(is, 13)];
			      }
			      T2g = T1Q - T1R;
			      T1S = T1Q + T1R;
			 }
		    }
	       }
	       {
		    E T1P, T25, T23, T2h, T1W, T1y, TS, T1z, TX, T27, T2a;
		    {
			 E Tf, Tu, T29, T28;
			 {
			      E T1U, TQ, T1V, TR, TW;
			      T1P = T7 - Te;
			      Tf = T7 + Te;
			      T1U = TO + TP;
			      TQ = TO - TP;
			      Tu = Tm + Tt;
			      T25 = Tt - Tm;
			      T23 = T1Z - T22;
			      T29 = T1Z + T22;
			      T2h = T1U - T1T;
			      T1V = T1T + T1U;
			      TR = TN + TQ;
			      TW = TN - TQ;
			      T27 = Tf + Tu;
			      T1W = T1S - T1V;
			      T28 = T1S + T1V;
			      T1y = FNMS(KP707106781, TR, TK);
			      TS = FMA(KP707106781, TR, TK);
			      T1z = FNMS(KP707106781, TW, TV);
			      TX = FMA(KP707106781, TW, TV);
			      T2a = T28 + T29;
			 }
			 ro[WS(ros, 8)] = Tf - Tu;
			 io[WS(ios, 8)] = T29 - T28;
		    }
		    ro[0] = T27 + T2a;
		    ro[WS(ros, 16)] = T27 - T2a;
		    {
			 E T2s, T2i, T2v, T2f, T2r, T2p, T2l, T2t;
			 {
			      E T2o, T2e, T26, T24;
			      T2o = T2d - T2c;
			      T2e = T2c + T2d;
			      T2s = FNMS(KP414213562, T2g, T2h);
			      T2i = FMA(KP414213562, T2h, T2g);
			      T26 = T23 - T1W;
			      T24 = T1W + T23;
			      T2v = FNMS(KP707106781, T2e, T2b);
			      T2f = FMA(KP707106781, T2e, T2b);
			      T2r = FMA(KP707106781, T2o, T2n);
			      T2p = FNMS(KP707106781, T2o, T2n);
			      io[WS(ios, 4)] = FMA(KP707106781, T26, T25);
			      io[WS(ios, 12)] = FMS(KP707106781, T26, T25);
			      ro[WS(ros, 4)] = FMA(KP707106781, T24, T1P);
			      ro[WS(ros, 12)] = FNMS(KP707106781, T24, T1P);
			      T2l = FNMS(KP414213562, T2k, T2j);
			      T2t = FMA(KP414213562, T2j, T2k);
			 }
			 {
			      E T1v, T1G, TH, T1s, T1F, T1w, T1o, T1g, T1p, T1n;
			      {
				   E T1f, TY, T1t, T1u, T1j, T1m;
				   {
					E Tz, TG, T1q, T1r;
					T1v = FNMS(KP707106781, Ty, Tv);
					Tz = FMA(KP707106781, Ty, Tv);
					{
					     E T2q, T2m, T2w, T2u;
					     T2q = T2l - T2i;
					     T2m = T2i + T2l;
					     T2w = T2t - T2s;
					     T2u = T2s + T2t;
					     io[WS(ios, 10)] = FMA(KP923879532, T2q, T2p);
					     io[WS(ios, 6)] = FMS(KP923879532, T2q, T2p);
					     ro[WS(ros, 2)] = FMA(KP923879532, T2m, T2f);
					     ro[WS(ros, 14)] = FNMS(KP923879532, T2m, T2f);
					     ro[WS(ros, 10)] = FNMS(KP923879532, T2w, T2v);
					     ro[WS(ros, 6)] = FMA(KP923879532, T2w, T2v);
					     io[WS(ios, 2)] = FMA(KP923879532, T2u, T2r);
					     io[WS(ios, 14)] = FMS(KP923879532, T2u, T2r);
					     TG = TC + TF;
					     T1G = TF - TC;
					}
					T1f = FNMS(KP198912367, T1e, T19);
					T1q = FMA(KP198912367, T19, T1e);
					T1r = FMA(KP198912367, TS, TX);
					TY = FNMS(KP198912367, TX, TS);
					T1t = FNMS(KP923879532, TG, Tz);
					TH = FMA(KP923879532, TG, Tz);
					T1u = T1r + T1q;
					T1s = T1q - T1r;
					T1F = FMA(KP707106781, T1i, T1h);
					T1j = FNMS(KP707106781, T1i, T1h);
					T1m = T1k + T1l;
					T1w = T1k - T1l;
				   }
				   ro[WS(ros, 7)] = FMA(KP980785280, T1u, T1t);
				   T1o = T1f - TY;
				   T1g = TY + T1f;
				   T1p = FMA(KP923879532, T1m, T1j);
				   T1n = FNMS(KP923879532, T1m, T1j);
				   ro[WS(ros, 9)] = FNMS(KP980785280, T1u, T1t);
			      }
			      ro[WS(ros, 1)] = FMA(KP980785280, T1g, TH);
			      ro[WS(ros, 15)] = FNMS(KP980785280, T1g, TH);
			      io[WS(ios, 1)] = FMS(KP980785280, T1s, T1p);
			      io[WS(ios, 15)] = FMA(KP980785280, T1s, T1p);
			      io[WS(ios, 9)] = FMS(KP980785280, T1o, T1n);
			      io[WS(ios, 7)] = FMA(KP980785280, T1o, T1n);
			      {
				   E T1A, T1D, T1N, T1O, T1K, T1L;
				   T1A = FMA(KP668178637, T1z, T1y);
				   T1K = FNMS(KP668178637, T1y, T1z);
				   T1L = FNMS(KP668178637, T1B, T1C);
				   T1D = FMA(KP668178637, T1C, T1B);
				   T1N = FNMS(KP923879532, T1w, T1v);
				   T1x = FMA(KP923879532, T1w, T1v);
				   T1O = T1K + T1L;
				   T1M = T1K - T1L;
				   ro[WS(ros, 5)] = FNMS(KP831469612, T1O, T1N);
				   T1I = T1D - T1A;
				   T1E = T1A + T1D;
				   T1J = FMA(KP923879532, T1G, T1F);
				   T1H = FNMS(KP923879532, T1G, T1F);
				   ro[WS(ros, 11)] = FMA(KP831469612, T1O, T1N);
			      }
			 }
		    }
	       }
	  }
	  io[WS(ios, 3)] = FMA(KP831469612, T1M, T1J);
	  ro[WS(ros, 3)] = FMA(KP831469612, T1E, T1x);
	  io[WS(ios, 13)] = FMS(KP831469612, T1M, T1J);
	  ro[WS(ros, 13)] = FNMS(KP831469612, T1E, T1x);
	  io[WS(ios, 11)] = FMA(KP831469612, T1I, T1H);
	  io[WS(ios, 5)] = FMS(KP831469612, T1I, T1H);
}

extern "C"
void launch(float *__restrict__ mem, hipStream_t stream) {
    dim3 grid(1, 1);
    dim3 threads(1, 1);

    r2hc_32 <<< grid, threads, 0, stream >>> (
        mem, &mem[32], &mem[33]);
}

